#include "hip/hip_runtime.h"
#include "definition.h"

__global__ void A2AsKernel(double *A, hipfftDoubleComplex *A00, hipfftDoubleComplex *A01, hipfftDoubleComplex *A02, hipfftDoubleComplex *A11, hipfftDoubleComplex *A12, hipfftDoubleComplex *A22, int NxFFT, int NyFFT, int NzFFT){
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int k = threadIdx.z + blockIdx.z*blockDim.z;
    int index = k + j*NzFFT + i*NzFFT*NyFFT;
    
    if(i < NxFFT && j <NyFFT && k<NzFFT){
        A00[index].x = A[0+2*0+12*index];
        A00[index].y = A[1+2*0+12*index];

        A01[index].x = A[0+2*1+12*index];
        A01[index].y = A[1+2*1+12*index];

        A02[index].x = A[0+2*2+12*index];
        A02[index].y = A[1+2*2+12*index];

        A11[index].x = A[0+2*3+12*index];
        A11[index].y = A[1+2*3+12*index];

        A12[index].x = A[0+2*4+12*index];
        A12[index].y = A[1+2*4+12*index];

        A22[index].x = A[0+2*5+12*index];
        A22[index].y = A[1+2*5+12*index];
    }
    

}

void A2As(double *A, hipfftDoubleComplex *A00, hipfftDoubleComplex *A01, hipfftDoubleComplex *A02, hipfftDoubleComplex *A11, hipfftDoubleComplex *A12, hipfftDoubleComplex *A22, int NxFFT, int NyFFT, int NzFFT){

    int tmpx = 10;
    int tmpy = 10;
    int tmpz = 10;
    dim3 dimBlock(tmpx, tmpy, tmpz);
    dim3 dimGrid(ceil((double)NxFFT/tmpx), ceil((double)NyFFT/tmpy), ceil((double)NzFFT/tmpz));   

    A2AsKernel<<<dimGrid, dimBlock>>>(A, A00, A01, A02, A11, A12, A22, NxFFT, NyFFT, NzFFT);

}

__global__ void B2BsKernel(double *bDev, hipfftDoubleComplex *bxDev, hipfftDoubleComplex *byDev, hipfftDoubleComplex *bzDev, int NxFFT, int NyFFT, int NzFFT){
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int k = threadIdx.z + blockIdx.z*blockDim.z;
    int index = k + j*NzFFT + i*NzFFT*NyFFT;
    
    if(i < NxFFT && j <NyFFT && k<NzFFT){
        bxDev[index].x = bDev[0+2*0+6*index];
        bxDev[index].y = bDev[1+2*0+6*index];

        byDev[index].x = bDev[0+2*1+6*index];
        byDev[index].y = bDev[1+2*1+6*index];

        bzDev[index].x = bDev[0+2*2+6*index];
        bzDev[index].y = bDev[1+2*2+6*index];
    }
    

}

void B2Bs(double *bDev, hipfftDoubleComplex *bxDev, hipfftDoubleComplex *byDev, hipfftDoubleComplex *bzDev, int NxFFT, int NyFFT, int NzFFT){
    int tmpx = 10;
    int tmpy = 10;
    int tmpz = 10;
    dim3 dimBlock(tmpx, tmpy, tmpz);
    dim3 dimGrid(ceil((double)NxFFT/tmpx), ceil((double)NyFFT/tmpy), ceil((double)NzFFT/tmpz));   

    B2BsKernel<<<dimGrid, dimBlock>>>(bDev, bxDev, byDev, bzDev, NxFFT, NyFFT, NzFFT);
}

__global__ void ConvKernel(hipfftDoubleComplex *Convx, hipfftDoubleComplex *Convy, hipfftDoubleComplex *Convz, hipfftDoubleComplex *A00, hipfftDoubleComplex *A01, hipfftDoubleComplex *A02, hipfftDoubleComplex *A11, hipfftDoubleComplex *A12, hipfftDoubleComplex *A22, hipfftDoubleComplex *bxDev, hipfftDoubleComplex *byDev, hipfftDoubleComplex *bzDev, int NxFFT, int NyFFT, int NzFFT){
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int k = threadIdx.z + blockIdx.z*blockDim.z;
    int index = k + j*NzFFT + i*NzFFT*NyFFT;
    
    if(i < NxFFT && j <NyFFT && k<NzFFT){
        Convx[index] = hipCadd(hipCadd(hipCmul(A00[index], bxDev[index]), hipCmul(A01[index], byDev[index])), hipCmul(A02[index], bzDev[index]));
        Convy[index] = hipCadd(hipCadd(hipCmul(A01[index], bxDev[index]), hipCmul(A11[index], byDev[index])), hipCmul(A12[index], bzDev[index]));
        Convz[index] = hipCadd(hipCadd(hipCmul(A02[index], bxDev[index]), hipCmul(A12[index], byDev[index])), hipCmul(A22[index], bzDev[index]));
    }
}
    
void Conv(hipfftDoubleComplex *Convx, hipfftDoubleComplex *Convy, hipfftDoubleComplex *Convz, hipfftDoubleComplex *A00, hipfftDoubleComplex *A01, hipfftDoubleComplex *A02, hipfftDoubleComplex *A11, hipfftDoubleComplex *A12, hipfftDoubleComplex *A22, hipfftDoubleComplex *bxDev, hipfftDoubleComplex *byDev, hipfftDoubleComplex *bzDev, int NxFFT, int NyFFT, int NzFFT){
    int tmpx = 10;
    int tmpy = 10;
    int tmpz = 10;
    dim3 dimBlock(tmpx, tmpy, tmpz);
    dim3 dimGrid(ceil((double)NxFFT/tmpx), ceil((double)NyFFT/tmpy), ceil((double)NzFFT/tmpz));   

    ConvKernel<<<dimGrid, dimBlock>>>(Convx, Convy, Convz, A00, A01, A02, A11, A12, A22, bxDev, byDev, bzDev, NxFFT, NyFFT, NzFFT);
}

__global__ void Conv2BKernel(hipfftDoubleComplex *Convx, hipfftDoubleComplex *Convy, hipfftDoubleComplex *Convz, double *bDev, int NxFFT, int NyFFT, int NzFFT){
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int k = threadIdx.z + blockIdx.z*blockDim.z;
    int index = k + j*NzFFT + i*NzFFT*NyFFT;
    
    if(i < NxFFT && j <NyFFT && k<NzFFT){
        bDev[0+2*0+6*index] = Convx[index].x;
        bDev[1+2*0+6*index] = Convx[index].y;

        bDev[0+2*1+6*index] = Convy[index].x;
        bDev[1+2*1+6*index] = Convy[index].y;

        bDev[0+2*2+6*index] = Convz[index].x;
        bDev[1+2*2+6*index] = Convz[index].y;
    }
    

}

void Conv2B(hipfftDoubleComplex *Convx, hipfftDoubleComplex *Convy, hipfftDoubleComplex *Convz, double *bDev, int NxFFT, int NyFFT, int NzFFT){
    int tmpx = 10;
    int tmpy = 10;
    int tmpz = 10;
    dim3 dimBlock(tmpx, tmpy, tmpz);
    dim3 dimGrid(ceil((double)NxFFT/tmpx), ceil((double)NyFFT/tmpy), ceil((double)NzFFT/tmpz));   

    Conv2BKernel<<<dimGrid, dimBlock>>>(Convx, Convy, Convz, bDev, NxFFT, NyFFT, NzFFT);
}

__global__ void APtoESumKernel(hipfftDoubleComplex *A00, hipfftDoubleComplex *A01, hipfftDoubleComplex *A02, hipfftDoubleComplex *A11, hipfftDoubleComplex *A12, hipfftDoubleComplex *A22, 
    hipfftDoubleComplex *PxDev, hipfftDoubleComplex *PyDev, hipfftDoubleComplex *PzDev,
    hipfftDoubleComplex *ESumxDev, hipfftDoubleComplex *ESumyDev, hipfftDoubleComplex *ESumzDev, 
    int NxFFT, int NyFFT, int NzFFT,
    int NxA, int NyA, int NzA, 
    int index1, int index2, int index3, int deduction){
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int k = threadIdx.z + blockIdx.z*blockDim.z;
    int index = k + j*NzFFT + i*NzFFT*NyFFT;
    int indexA = (k+index3) + (j+index2)*NzA +(i+index1)*NzA*NyA;
    if(i < NxFFT && j <NyFFT && k<NzFFT){
        if(deduction == 1){
            ESumxDev[index] = hipCadd(ESumxDev[index], hipCadd(hipCadd(hipCmul(A00[indexA], PxDev[0]), hipCmul(A01[indexA], PyDev[0])), hipCmul(A02[indexA], PzDev[0])));
            ESumyDev[index] = hipCadd(ESumyDev[index], hipCadd(hipCadd(hipCmul(A01[indexA], PxDev[0]), hipCmul(A11[indexA], PyDev[0])), hipCmul(A12[indexA], PzDev[0])));
            ESumzDev[index] = hipCadd(ESumzDev[index], hipCadd(hipCadd(hipCmul(A02[indexA], PxDev[0]), hipCmul(A12[indexA], PyDev[0])), hipCmul(A22[indexA], PzDev[0])));
        }
        if(deduction == -1){
            ESumxDev[index] = hipCsub(ESumxDev[index], hipCadd(hipCadd(hipCmul(A00[indexA], PxDev[0]), hipCmul(A01[indexA], PyDev[0])), hipCmul(A02[indexA], PzDev[0])));
            ESumyDev[index] = hipCsub(ESumyDev[index], hipCadd(hipCadd(hipCmul(A01[indexA], PxDev[0]), hipCmul(A11[indexA], PyDev[0])), hipCmul(A12[indexA], PzDev[0])));
            ESumzDev[index] = hipCsub(ESumzDev[index], hipCadd(hipCadd(hipCmul(A02[indexA], PxDev[0]), hipCmul(A12[indexA], PyDev[0])), hipCmul(A22[indexA], PzDev[0])));
        }
    }

}

void APtoESum(hipfftDoubleComplex *A00, hipfftDoubleComplex *A01, hipfftDoubleComplex *A02, hipfftDoubleComplex *A11, hipfftDoubleComplex *A12, hipfftDoubleComplex *A22, 
    hipfftDoubleComplex *PxDev, hipfftDoubleComplex *PyDev, hipfftDoubleComplex *PzDev,
    hipfftDoubleComplex *ESumxDev, hipfftDoubleComplex *ESumyDev, hipfftDoubleComplex *ESumzDev, int NxFFT, int NyFFT, int NzFFT, 
    int NxA, int NyA, int NzA,
    int index1, int index2, int index3, int deduction){
        int tmpx = 10;
        int tmpy = 10;
        int tmpz = 10;
        dim3 dimBlock(tmpx, tmpy, tmpz);
        dim3 dimGrid(ceil((double)NxFFT/tmpx), ceil((double)NyFFT/tmpy), ceil((double)NzFFT/tmpz));   

        APtoESumKernel<<<dimGrid, dimBlock>>>(A00, A01, A02, A11, A12, A22, PxDev, PyDev, PzDev, ESumxDev, ESumyDev, ESumzDev, NxFFT, NyFFT, NzFFT, NxA, NyA, NzA, index1, index2, index3, deduction);

}